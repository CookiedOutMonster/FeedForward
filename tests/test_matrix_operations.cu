#include <hip/hip_runtime.h>
#include <gtest/gtest.h>

#include "MatrixOperations.h"

TEST(DotProduct, DotProduct_5vector_x_5vector_CPU) {
    // Dot product of two 5x1 vectors, output should be 70

    int expectedValue = 70;

    double vector_1[5] = {1, 2, 3, 4, 5};
    double vector_2[5] = {2, 3, 4, 5, 6};
    int size = 5;

    int actualValue = DotProduct(vector_1, vector_2, size);

    ASSERT_EQ(expectedValue, actualValue);
}

TEST(DotProduct, DotProduct_5vector_x_5vector_GPU) {
    int expectedValue = 70;

    double vector_1[5] = {1, 2, 3, 4, 5};
    double vector_2[5] = {2, 3, 4, 5, 6};
    int size = 5;

    // Allocate memory on the device
    double *d_vector_1, *d_vector_2;
    int *d_result;
    hipMalloc(&d_vector_1, size * sizeof(double));
    hipMalloc(&d_vector_2, size * sizeof(double));
    hipMalloc(&d_result, sizeof(int));

    // Copy data from host to device
    hipMemcpy(d_vector_1, vector_1, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_vector_2, vector_2, size * sizeof(double), hipMemcpyHostToDevice);

    // Launch the kernel (using 1 thread block with 1 thread)
    DotProductKernel<<<1, 1>>>(d_vector_1, d_vector_2, d_result, size);

    // Copy the result from device to host
    int actualValue;
    hipMemcpy(&actualValue, d_result, sizeof(int), hipMemcpyDeviceToHost);

    // Free the device memory
    hipFree(d_vector_1);
    hipFree(d_vector_2);
    hipFree(d_result);

    // Assert the result is as expected
    ASSERT_EQ(expectedValue, actualValue);
}
